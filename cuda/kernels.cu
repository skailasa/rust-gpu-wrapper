#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(err)); \
        return; \
    }

extern "C" void cuda_add_kernel(float *a, float *b, float *c, int n);

__global__ void add(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

extern "C" void cuda_add_kernel(float *a, float *b, float *c, int n) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    float *a_d;
    float *b_d;
    float *c_d;
    CUDA_CHECK(hipMalloc(&a_d, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&b_d, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&c_d, n * sizeof(float)));
    CUDA_CHECK(hipMemset(c_d, 0, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(a_d, a, n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(b_d, b, n * sizeof(float), hipMemcpyHostToDevice));

    add<<<blocks, threads>>>(a_d, b_d, c_d, n);
    CUDA_CHECK(hipDeviceSynchronize());  // wait for kernel to finish

    CUDA_CHECK(hipMemcpy(c, c_d, n * sizeof(float), hipMemcpyDeviceToHost));
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    auto err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }
}
